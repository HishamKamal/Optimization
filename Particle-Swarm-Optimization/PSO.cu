#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include <hipcub/hipcub.hpp>

#include "PSO.cuh"
#include "Utilities.cuh"
#include "Functional.cuh"

#define BLOCKSIZE_LOCAL_BEST   256

#define DEBUG

texture<float, 1, hipReadModeElementType> functional_texture;

/******************/
/* PSO PARAMETERS */
/******************/
#define numberOfParticles		10							// --- Number of individuals
#define numberOfUnknowns		 5							// --- Number of unknowns
#define numberOfGenerations    100							// --- Number of iterations
#define RADIUS					 2							// --- Radius of the ring neighboorhood to which each particle communicates
#define W						 0.7213475204444817278f		// --- Inertia weight W
#define C1						 1.193147180559945286f		// --- Cognitive attraction factor C1
#define C2						 1.193147180559945286f		// --- Social attraction factor C2
#define BLOCKSIZE				 (numberOfUnknowns)

/********************/
/* GLOBAL VARIABLES */
/********************/
hiprandState		*devStates; 
unsigned int	actualParticleSize;
unsigned int	*d_localBestIDs; 
unsigned int	*d_to_be_updated; 
int				vectorSwarmSize;
int				scalarFloatSwarmSize;
int				scalarIntSwarmSize;
float			h_globalBestFitness;					// --- Final global best fitness value
float			*h_globalBestPosition;
float			*d_positions; 
float			*d_best_personal_positions; 
float			*d_velocities; 
float			*d_functionals; 
float			*d_personal_best_functional; 

// --- Lower limit for each dimension of the search space MIN_VALS
const float MINVALS[numberOfUnknowns] = {-5.12f, -5.12f, -5.12f, -5.12f, -5.12f};
// --- Upper limit for each dimension of the search space MAX_VALS
const float MAXVALS[numberOfUnknowns] = { 5.12f,  5.12f,  5.12f,  5.12f,  5.12f};

float DELTAVALS[numberOfUnknowns];

/************************/
/* CONSTANT MEMORY DATA */
/************************/
// --- Starting coordinates of the hypercubical search space 
__constant__ float  c_minValues[numberOfUnknowns];

// --- Ending coordinates of the hypercubical search space 
__constant__ float  c_maxValues[numberOfUnknowns];

// --- Widths of the hypercubical search space 
__constant__ float  c_deltaValues[numberOfUnknowns];

/*********************************************/
/* INITIALIZATION OF RANDOM NUMBER GENERATOR */
/*********************************************/
__global__ void init_pseudorandom_generator(hiprandState * __restrict__ state, const unsigned long seed)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, id, 0, &state[id]);
}

/********************************************************/
/* INITIALIZATIONS OF PARTICLE POSITIONS AND VELOCITIES */
/********************************************************/
__global__ void particles_initialization(float * __restrict__ d_positions,  float * __restrict__ d_best_personal_positions, 
	                                     float * __restrict__ d_velocities, hiprandState * __restrict__ devStates) {

	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// --- Position initializations
	float R = hiprand_uniform(&devStates[tid]);
	float pos = c_minValues[threadIdx.x] + R * c_deltaValues[threadIdx.x];
	d_positions[tid]     = pos;
	d_best_personal_positions[tid] = pos;

	// --- Velocity initializations
	R = hiprand_uniform(&devStates[tid]);
	float vel = c_minValues[threadIdx.x] + R * c_deltaValues[threadIdx.x];
	d_velocities[tid] = (vel - pos) / 2.0;
}

/**********************/
/* PSO INITIALIZATION */
/**********************/
void h_PSO_Initialize() {

	// To achieve byte alignment, we need data arrays with a number of elements for each particle which is a multiple of 16
	//unsigned int actualParticleSize = iAlignUp(numberOfUnknowns, 16);
	actualParticleSize = numberOfUnknowns;

	vectorSwarmSize = numberOfParticles * actualParticleSize * sizeof(float);

	// --- Initializing the positions (numberOfParticles * actualParticleSize)
	gpuErrchk(hipMalloc((void**)&d_positions, vectorSwarmSize));
	gpuErrchk(hipMemset(d_positions, 0, vectorSwarmSize));

	// --- Allocation of the current personal best positions array (numberOfParticles * actualParticleSize)
	gpuErrchk(hipMalloc((void**)&d_best_personal_positions, vectorSwarmSize));
	gpuErrchk(hipMemset(d_best_personal_positions, 0, vectorSwarmSize));

	// --- Initializing the current velocities (numberOfParticles * actualParticleSize)
	gpuErrchk(hipMalloc((void**)&d_velocities, vectorSwarmSize));
	gpuErrchk(hipMemset(d_velocities, 0, vectorSwarmSize));

	scalarFloatSwarmSize = numberOfParticles * sizeof(float);

	// --- Initializing fitnesses (numberOfParticles) and binding the texture
	size_t pitch;
	gpuErrchk(hipMallocPitch(&d_functionals, &pitch, scalarFloatSwarmSize, 1));
	gpuErrchk(hipMemset(d_functionals, 0, scalarFloatSwarmSize));

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	size_t texture_offset = 0;
	gpuErrchk(hipBindTexture2D(&texture_offset, functional_texture, d_functionals, channelDesc, numberOfParticles, 1, pitch)); 
    functional_texture.normalized = true; 
    functional_texture.addressMode[0] = hipAddressModeWrap;

	// --- Initializing the current personal best fitnesses (numberOfParticles)
	gpuErrchk(hipMalloc((void**)&d_personal_best_functional, scalarFloatSwarmSize));
	gpuErrchk(hipMemset(d_personal_best_functional, 0, scalarFloatSwarmSize));

	scalarIntSwarmSize = numberOfParticles * sizeof(unsigned int);
	
	// --- Allocation of the local best ids, namely, the indices (for all particles) of the best neighbour (for the ring topology) (numberOfParticles)
	gpuErrchk(hipMalloc((void**)&d_localBestIDs, scalarIntSwarmSize));
	gpuErrchk(hipMemset(d_localBestIDs, 0, scalarIntSwarmSize));

	// --- Allocation of the update flags saying to each particle whether to update their personal best (numberOfParticles)
	gpuErrchk(hipMalloc((void**)&d_to_be_updated, scalarIntSwarmSize));
	gpuErrchk(hipMemset(d_to_be_updated, 0, scalarIntSwarmSize));

	// --- Allocation of the CUDA random states
	gpuErrchk(hipMalloc((void **)&devStates, numberOfParticles * actualParticleSize * sizeof(hiprandState)));

	// --- Setting up random generator states
	init_pseudorandom_generator<<<numberOfParticles, actualParticleSize>>>(devStates, time(NULL));
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	h_globalBestPosition = (float *)malloc(numberOfUnknowns * sizeof(float));

}
	
	
/************************************/
/* UPDATE OF THE PARTICLE POSITIONS */
/************************************/
__global__ void g_positionsUpdate(const unsigned int * __restrict__ d_localBestIDs, 
	                                     const unsigned int * __restrict__ d_to_be_updated, float * __restrict__ d_positions, 
										 float * __restrict__ d_best_personal_positions, float * __restrict__ d_velocities, 
										 hiprandState * __restrict__ devStates){

	// --- blockIdx.x addresses the particle inside the swarm
	// --- threadIdx.x addresses the dimension (unknown) within the particle

	// --- gridDim.x represents the NUMBER_OF_PARTICLES
	// --- blockDim.x represents the actualParticleSize (PROBLEM_DIMENSIONS aligned up to a multiple of 16)

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int s_update;		// --- Best position update flag (tells if the specific particle is the global best or not)
	__shared__ unsigned int s_bestID; 		// --- Beginning of the local best individual

	//The first thread load the best position s_update flag and the index of the local best individual
	if(threadIdx.x == 0){
		s_update = d_to_be_updated[blockIdx.x];
		s_bestID = d_localBestIDs[blockIdx.x] * blockDim.x;
	}
	__syncthreads();

	float pos		= d_positions[tid];			// --- Current position
	float bestPos	= d_best_personal_positions[tid];		// --- Current particle best position
	float vel		= d_velocities[tid];		// --- Current velocity

	// --- Load the 2 pseudo random numbers needed to update position and velocity
	float R1, R2;
	R1 = hiprand_uniform(&devStates[tid]);
	R2 = hiprand_uniform(&devStates[tid]);

	if (s_update){
		bestPos = pos;										// --- Update local best position
		d_best_personal_positions[tid] = bestPos;			// --- Update the local best position to global memory
	}

	__threadfence(); 
	
	vel *= W;														// --- Apply inertia factor
	vel += C1 * R1 * (bestPos - pos);								// --- Add to the velocity the Cognitive Contribution
	
	vel += C2 * R2 * (d_best_personal_positions[s_bestID+threadIdx.x] - pos);	// --- Add to the velocity the Social Contribution

	d_velocities[tid] = vel;										// --- Update velocity to global memory

	pos += vel;														// --- Position update

	// --- Clamping the position to the actual search space
	pos = min(pos, c_maxValues[threadIdx.x]);
	pos = max(pos, c_minValues[threadIdx.x]);

	d_positions[tid] = pos;											// --- Update position to global memory
}

/***********************************/
/* FIND THE LOCAL BEST PARTICLE ID */
/***********************************/
__global__ void local_best_update(const float * __restrict__ d_functionals, float * __restrict__ d_personal_best_functional, 
	                              unsigned int * __restrict__ d_localBestIDs, unsigned int * __restrict__ d_to_be_updated, const unsigned int generationNumber){

	// --- particleID addresses the particle inside the swarm
	// --- blockDim.x * gridDim.x is greater or equal to all the particles
	int particleID = threadIdx.x + blockIdx.x * blockDim.x;

	if (particleID < numberOfParticles) {
	
		float *local_functionals = (float *)malloc((2 * RADIUS + 1) * sizeof(float));

		// --- Load the functional values from global memory
		for (int i = 0; i < 2 * RADIUS + 1; i++) local_functionals[i] = tex1D(functional_texture, (float)(particleID + 0.5 + (i - RADIUS)) / (float)numberOfParticles);

		thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(local_functionals);
		thrust::device_ptr<float> min_ptr = thrust::min_element(thrust::seq, dev_ptr, dev_ptr + 2 * RADIUS + 1);
		float min_value = min_ptr[0];

		// --- Writes the local best-ID to global memory
		d_localBestIDs[particleID] = (particleID + (&min_ptr[0] - &dev_ptr[0]) - RADIUS + numberOfParticles) % numberOfParticles;

		if (generationNumber > 0) {
			// --- Possibly update of both the best fitness value and the best position update flag
			unsigned int update = local_functionals[RADIUS] < d_personal_best_functional[particleID];
			d_to_be_updated[particleID] = update;
			if (update) d_personal_best_functional[particleID] = local_functionals[RADIUS]; 
		}
		else
			// --- Initially, the best personal fitness value is the first fitness value
			d_personal_best_functional[particleID] = local_functionals[RADIUS];
	}	
}

/*****************************************************************************/
/* FUNCTION TO FIND THE GLOBAL BEST PARTICLE AFTER COMPLETING THE ITERATIONS */
/*****************************************************************************/
void h_findGlobalBest(float * __restrict__ h_globalBestFitness, unsigned int * __restrict__ h_globalBestID, 
				      float * __restrict__ d_personal_best_functional){

	thrust::device_ptr<float> dp = thrust::device_pointer_cast(d_personal_best_functional);
	thrust::device_ptr<float> pos = thrust::min_element(dp, dp + numberOfParticles);

	*h_globalBestID = thrust::distance(dp, pos);

	gpuErrchk(hipMemcpy(h_globalBestFitness, &d_personal_best_functional[*h_globalBestID], sizeof(float), hipMemcpyDeviceToHost));
}

/******************************/
/* TRANSFORM REDUCTION KERNEL */
/******************************/
__global__ void CostFunctionalCalculation(const float * __restrict__ indata, float * __restrict__ outdata) {
	
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	// --- Specialize BlockReduce for type float. 
	typedef hipcub::BlockReduce<float, numberOfUnknowns> BlockReduceT;

	__shared__ typename BlockReduceT::TempStorage  temp_storage;

	float result;
	if(tid < (numberOfParticles * numberOfUnknowns) ) result = BlockReduceT(temp_storage).Sum(sphere(indata[tid]));

	if(threadIdx.x == 0) outdata[blockIdx.x] = result; 
  
	return;
}

/*************************/
/* OPTIMIZATION FUNCTION */
/*************************/
void h_PSO_Optimize(void)
{
	printf("Starting Optimization...\n");

	for (int i = 0; i < numberOfUnknowns; i++) DELTAVALS[i] = MAXVALS[i] - MINVALS[i];
	
	// --- Set up search space limits
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c_minValues),    &MINVALS,     numberOfUnknowns * sizeof(float), 0, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c_maxValues),    &MAXVALS,     numberOfUnknowns * sizeof(float), 0, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(c_deltaValues),  &DELTAVALS,   numberOfUnknowns * sizeof(float), 0, hipMemcpyHostToDevice));

	// --- Particles initialization
	particles_initialization<<<numberOfParticles, actualParticleSize>>>(d_positions, d_best_personal_positions, d_velocities, devStates);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	// --- Set the update flags to zero
	gpuErrchk(hipMemset(d_to_be_updated, 0, numberOfParticles * sizeof(unsigned int)));

	// --- First fitnesses evaluation
	CostFunctionalCalculation<<<numberOfParticles, numberOfUnknowns>>>(d_positions, d_functionals);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	unsigned int generationNumber = 0;
	
	// --- First time local bests update
	local_best_update<<<iDivUp(numberOfParticles, BLOCKSIZE_LOCAL_BEST), BLOCKSIZE_LOCAL_BEST>>>(d_functionals, d_personal_best_functional, d_localBestIDs, d_to_be_updated, generationNumber);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	// --- GENERATIONS
	for(generationNumber = 1; generationNumber < numberOfGenerations; ++generationNumber){

		// --- PositionS Update
		g_positionsUpdate<<<numberOfParticles, actualParticleSize>>>(d_localBestIDs, d_to_be_updated, d_positions, d_best_personal_positions, d_velocities, devStates);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// --- Fitness evaluation
		CostFunctionalCalculation<<<numberOfParticles, numberOfUnknowns>>>(d_positions, d_functionals);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// --- Local bests update
		local_best_update<<<iDivUp(numberOfParticles, BLOCKSIZE_LOCAL_BEST), BLOCKSIZE_LOCAL_BEST>>>(d_functionals, d_personal_best_functional, d_localBestIDs, d_to_be_updated, generationNumber);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

	}

	// --- Global best determination
	unsigned int h_globalBestID;
	h_findGlobalBest(&h_globalBestFitness, &h_globalBestID, d_personal_best_functional);

	gpuErrchk(hipMemcpy(h_globalBestPosition, d_best_personal_positions + h_globalBestID * actualParticleSize, numberOfUnknowns * sizeof(float), hipMemcpyDeviceToHost));

	printf("Number of partices = %d\n Number of unknowns = %d\n Number of iterations = %d\n Minimum found = %e\n", numberOfParticles, numberOfUnknowns, numberOfGenerations, h_globalBestFitness);

	for (int k=0; k<numberOfUnknowns; k++) printf("%i %f\n", k, h_globalBestPosition[k]);
}
